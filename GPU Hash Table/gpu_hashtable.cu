#include "hip/hip_runtime.h"
// Calin Bucur 332CB
// Tema 3 ASC

#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

using namespace std;

// Number of threads in a block
#define BLOCK_SIZE 256

// Hashing function
// Got it from the internet
// Apparently the distribution is quite uniform for reasons beyond my understanding
__device__ int hash_func(int key) {
	int hash = key;
	hash = ((hash >> 16) ^ hash) * 0x45d9f3b;
    hash = ((hash >> 16) ^ hash) * 0x45d9f3b;
    hash = (hash >> 16) ^ hash;
    return hash;
}

// Kernel that rehashes every key-value pair from the old table and inserts it into the new one
__global__ void rehash (Data *old_arr, Data *new_arr, int old_size, int new_size) {
	// Get the index of the key-value pair the current thread should rehash
	unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	// Check if the index is within bounds and if there is a pair at that index
	if (idx < old_size && old_arr[idx].key != 0) {
		// Compute the new hash
		int hash = hash_func(old_arr[idx].key);
		// Get the position where it should be inserted
		hash %= new_size;
		// Atomically check if the position is free and insert the key
		atomicCAS(&new_arr[hash].key, 0, old_arr[idx].key);
		// Check if the key was inserted
		if (new_arr[hash].key == old_arr[idx].key) {
			// Insert the value
			new_arr[hash].value = old_arr[idx].value;
		} else { // Liniar probing
			// Get the next posible position
			hash++;
			// Loop until the pair was inserted
			// We have the guarantee the loop will break at some point
			while(1) {
				// Atomically check if the position is free and insert the key
				atomicCAS(&new_arr[hash].key, 0, old_arr[idx].key);
				// Check if the key was inserted
				if (new_arr[hash].key == old_arr[idx].key) {
					// Insert the value
					new_arr[hash].value = old_arr[idx].value;
					// Break the loop
					return;
				} else {
					// Move to the next hash
					// If the end of the table was reached start from 0
					hash = (hash + 1) % new_size;
				}
			}
		}
	}
}

// Kernel that inserts a batch of key-value pairs in the table
// Uses liniar probing for collisions
__global__ void insert(Data *arr, int capacity, int *keys, int *values, int numKeys, int *existing) {
	// Get the index of the key-value pair the current thread should insert
	unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	// Check if the index is in bounds
	if (idx < numKeys) {
		Data pair;
		pair.key = keys[idx];
		pair.value = values[idx];
		// Compute the hash
		int hash = hash_func(pair.key);
		// Get the position
		hash %= capacity;
		// Check if the key is already inserted at that position
		if (arr[hash].key == pair.key) {
			// Update the value
			arr[hash].value = pair.value;
			// Decrement the number of pairs inserted
			atomicSub(existing, 1); // Didn't use atomicDec because it had a weird condition
		}
		else {
			// Atomically check if the position is free and insert the key
			atomicCAS(&arr[hash].key, 0, pair.key);
			if (arr[hash].key == pair.key) {
				// Insert the value
				arr[hash].value = pair.value;
			}
			else { // Liniar probing
				// Get the next possible position
				hash++;
				// Loop until the pair was inserted
				// We have the guarantee the loop will break at some point
				while(1) {
					// If the key a;ready exists at that position
					if (arr[hash].key == pair.key) {
						// Update the value
						arr[hash].value = pair.value;
						// Decrement the number of pairs inserted
						atomicSub(existing, 1);
						return;
					} else {
						// Atomically check if the position is free and insert the key
						atomicCAS(&arr[hash].key, 0, pair.key);
						// Check if the key was inserted
						if (arr[hash].key == pair.key) {
							// Insert the value
							arr[hash].value = pair.value;
							return;
						}
						else {
							// Go to the next position
							hash = (hash + 1) % capacity;
						}
					}
				}
			}
		}
	}
}

// Kernel that gets the values for a batch of keys
__global__ void get(Data *arr, int *keys, int *values, int capacity, int numKeys) {
	// Get the index of the key the current thread should look for
	unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	// Check that the index is in bounds
	if (idx < numKeys) {
		int key = keys[idx]; // Get the key
		// Compute the hash
		int hash = hash_func(key);
		// Get the position
		hash %= capacity;
		// Check if the key is there
		if (arr[hash].key == key) {
			// Get the value
			values[idx] = arr[hash].value;
		}
		else { // Liniar probing
			hash++;
			// Check element by element until the key is found
			while (1) {
				if (arr[hash].key == key) {
					values[idx] = arr[hash].value;
					break;
				}
				else {
					hash = (hash + 1) % capacity;
				}
			}
		}
	}
}

// Hash Table constructor
GpuHashTable::GpuHashTable(int size) {
	this->capacity = size;
	this->size = 0; // Initially the table is empty
	// Allocate the table in the VRAM
	glbGpuAllocator->_cudaMalloc((void **)&this->arr, this->capacity * sizeof(Data));
	cudaCheckError();
	// Set all the positions as empty (a.k.a zero)
	hipMemset(this->arr, 0, this->capacity * sizeof(Data));
}

// Hash Table destructor
GpuHashTable::~GpuHashTable() {
	glbGpuAllocator->_cudaFree(this->arr);
}

// Resizes the table
void GpuHashTable::reshape(int numBucketsReshape) {
	Data *new_arr;
	// Allocate the new array of the desired size
	glbGpuAllocator->_cudaMalloc((void **)&new_arr, numBucketsReshape * sizeof(Data));
	cudaCheckError();
	// Initialize all the positions as empty
	hipMemset(new_arr, 0, numBucketsReshape * sizeof(Data));
	cudaCheckError();
	// Calculate the number of blocks necessary
	int block_num = this->capacity / BLOCK_SIZE;
	if (this->capacity % BLOCK_SIZE)
		block_num++;
	// Call the kernel
	rehash<<<block_num, BLOCK_SIZE>>>(this->arr, new_arr, this->capacity, numBucketsReshape);
	hipDeviceSynchronize();
	cudaCheckError();
	// Free the old table
	glbGpuAllocator->_cudaFree(this->arr);
	cudaCheckError();
	// Assign the new table
	this->arr = new_arr;
	// Update the capacity
	this->capacity = numBucketsReshape;
}

// Inserts a batch of key-value pairs
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	// If the capacity would be exceeded
	if (this->size + numKeys >= this->capacity) {
		// Double the capacity
		this->reshape((this->size + numKeys) * 2);
	}
	// Calculate the number of blocks necessary
	int block_num = numKeys / BLOCK_SIZE;
	if (numKeys % BLOCK_SIZE)
		block_num++;
	int *GPU_keys;
	int *GPU_values;
	int *GPU_numKeys;
	// Allocate arrays for the keys and values to be inserted
	glbGpuAllocator->_cudaMalloc((void **)&GPU_keys, numKeys * sizeof(int));
	cudaCheckError();
	glbGpuAllocator->_cudaMalloc((void **)&GPU_values, numKeys * sizeof(int));
	cudaCheckError();
	// Allocate a pointer for the number of keys inserted
	// This will be updated with the number of keys that were actually inserted (not updated)
	glbGpuAllocator->_cudaMalloc((void **)&GPU_numKeys, sizeof(int));
	cudaCheckError();
	// Copy the data into the GPU arrays
	hipMemcpy(GPU_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	cudaCheckError();
	hipMemcpy(GPU_values, values, numKeys * sizeof(int), hipMemcpyHostToDevice);
	cudaCheckError();
	hipMemcpy(GPU_numKeys, &numKeys, sizeof(int), hipMemcpyHostToDevice);
	cudaCheckError();
	// Call the kernel
	insert<<<block_num, BLOCK_SIZE>>>(this->arr, this->capacity, GPU_keys, GPU_values, numKeys, GPU_numKeys);
	hipDeviceSynchronize();
	cudaCheckError();
	// Get the number of inserted keys
	hipMemcpy(&numKeys, GPU_numKeys, sizeof(int), hipMemcpyDeviceToHost);
	cudaCheckError();
	// Update the size
	this->size += numKeys;
	// Free the gpu arrays
	glbGpuAllocator->_cudaFree(GPU_keys);
	cudaCheckError();
	glbGpuAllocator->_cudaFree(GPU_values);
	cudaCheckError();
	glbGpuAllocator->_cudaFree(GPU_numKeys);
	cudaCheckError();
	return true;
}

// Gets a batch of values corresponding to the given keys
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	// Calculate the necessary number of blocks
	int block_num = numKeys / BLOCK_SIZE;
	if (numKeys % BLOCK_SIZE)
		block_num++;
	int *GPU_keys;
	int *GPU_values;
	// Allocate GPU arrays for the keys and values
	glbGpuAllocator->_cudaMalloc((void **)&GPU_keys, numKeys * sizeof(int));
	cudaCheckError();
	glbGpuAllocator->_cudaMalloc((void **)&GPU_values, numKeys * sizeof(int));
	cudaCheckError();
	// Copy the keys
	hipMemcpy(GPU_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	cudaCheckError();
	// Call the kernel
	get<<<block_num, BLOCK_SIZE>>>(this->arr, GPU_keys, GPU_values, this->capacity, numKeys);
	hipDeviceSynchronize();
	cudaCheckError();
	// Allocate an array for the values
	int *values = (int*)malloc(numKeys * sizeof(int));
	// Copy the values from the GPU
	hipMemcpy(values, GPU_values, numKeys * sizeof(int), hipMemcpyDeviceToHost);
	cudaCheckError();
	// Free the GPU arrays
	glbGpuAllocator->_cudaFree(GPU_keys);
	cudaCheckError();
	glbGpuAllocator->_cudaFree(GPU_values);
	cudaCheckError();
	return values;
}
